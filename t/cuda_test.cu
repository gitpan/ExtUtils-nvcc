#include "hip/hip_runtime.h"
// This is a simple cuda program that should be easily compilable with nvcc, and
// thus perl_nvcc. Note that nvcc always invokes the C++ compiler, so we should
// use actual C++ compatible code instead of plain C code:

#include <cstdio>
#include <cstring>

// This is a kernel that 'fixes' fubarred text. The text was fubarred by adding
// 1 to each character; this subtracts 1 from each character.
__global__ void fix_kernel(char * text) {
	text[threadIdx.x] -= 1;
}

// Fubars the message by adding one to each character:
void fubar_message (char * message) {
	int i;
	for (i = 0; i < 11; i++) {
		message[i] += 1;
	}
}

// Creates memory on the device for the message and croaks on error:
char * create_dev_message () {
	// allocate memory on the device and check for errors:
	char * dev_message;
	hipError_t err = hipMalloc(&dev_message, 12);
	if (err != hipSuccess) {
		printf("Trouble with memory!\n");
		exit(1);
	}
	
	return dev_message;
}

// Copies the message to the device and croaks on error:
void copy_message_to_dev(char * message, char * dev_message) {
	hipError_t err = hipMemcpy(dev_message, message, 12, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("Trouble copying memory to the device!\n");
		exit(2);
	}
}

// Runs the kernel that's supposed to fix the text, and croaks on error:
void run_fix_kernel(char * dev_message) {
	// run the kernel on the device:
	fix_kernel<<<1, 11>>>(dev_message);
	// Check for errors:
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("Trouble running the kernel!\n");
		exit(3);
	}
}

// Copies the (unfubarred) message back to the host, and croaks on error:
void copy_message_to_host(char * message, char * dev_message) {
	hipError_t err = hipMemcpy(message, dev_message, 12, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("Trouble copying memory back to host!\n");
		exit(4);
	}
}

// Cleans up the memory on the device and croaks on error:
void clean_up_dev_memory(char * dev_message) {
	hipError_t err = hipFree(dev_message);
	if (err != hipSuccess) {
		printf("Trouble freeing device memory!\n");
		exit(5);
	}
}

// Tests that the resulting text is correct:
void test_result(char * message) {
	if (strncmp(message, "good to go!", 12) == 0) {
		printf("Success");
	}
	else {
		printf("%s", message);
	}
}

int main() {
	// This has 11 fidlable characters. The twelfth should not be fiddled:
	char message[12] = "good to go!";
	
	// mess up the message
	fubar_message(message);
	
	// Allocate the device memory and copy the contents:
	char * dev_message = create_dev_message();
	copy_message_to_dev(message, dev_message);
	
	// Run the kernel to fix the message:
	run_fix_kernel(dev_message);
	
	// Copy the result back and clean up the cuda memory:
	copy_message_to_host(message, dev_message);
	clean_up_dev_memory(dev_message);
	
	// Test that the message was restored:
	test_result(message);
}
